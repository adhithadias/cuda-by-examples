
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <time.h>
#include <stdlib.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__
void saxpy(int *x, int *y, int alpha, size_t N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<N) {
		y[i] = alpha * y[i] + x[i];
	}
}

void initialize_list(int *x, int N) {
	for (int i=0; i<N; i++) {
		x[i] = rand();
	}
}

int main (void) {
	int N = std::pow(10, 7);
	int alpha = 2;

	int *x, *y;
	x = (int *) (malloc(N * sizeof(int)));
	y = (int *) (malloc(N * sizeof(int)));
	
	srand(time(NULL));
	initialize_list(x, N);
	initialize_list(y, N);
	
	int *d_x, *d_y;
	hipMalloc((void **)&d_x, N*sizeof(int));
	hipMalloc((void **)&d_y, N*sizeof(int));
	
	hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);
	
	saxpy<<<(N+255)/256, 256>>>(d_x, d_y, alpha, N);
	
	int *c;
	c = (int *)(malloc(N*sizeof(int)));
	
	hipMemcpy(c, d_y, N*sizeof(int), hipMemcpyDeviceToHost);
	
	printf("[");
	for (int i=0; i<N; i++) {
		 
		if (i<10) {
			printf("%d ", c[i]);
		}
		
		if (y[i]*alpha + x[i] != c[i]) {
			printf("YOU SCREWED UP!");
		}
	}
	printf(" ... ]");
	
	hipFree(d_x);
	return 0;
	
}


